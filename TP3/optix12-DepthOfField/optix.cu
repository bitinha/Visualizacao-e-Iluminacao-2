#include "hip/hip_runtime.h"
#include <optix.h>
#include "random.h"
#include "LaunchParams7.h" // our launch params
#include <vec_math.h> // NVIDIAs math utils

extern "C" {
    __constant__ LaunchParams optixLaunchParams;
}

//  a single ray type
enum { PHONG=0, SHADOW, RAY_TYPE_COUNT };

struct colorPRD{
    float3 color;
    unsigned int seed;
} ;

struct shadowPRD{
    float shadowAtt;
    unsigned int seed;
} ;


// -------------------------------------------------------
// closest hit computes color based lolely on the triangle normal

extern "C" __global__ void __closesthit__radiance() {

    colorPRD &prd = *(colorPRD *)getPRD<colorPRD>();

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    // intersection position
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax()*optixGetWorldRayDirection();

    // direction towards light
    float3 lPos = make_float3(optixLaunchParams.global->lightPos);
    float3 lDir = normalize(lPos - pos);
    float3 nn = normalize(make_float3(n));
    float intensity = max(dot(lDir, nn),0.0f);
    

    // ray payload
    shadowPRD shadowAttPRD;
    shadowAttPRD.shadowAtt = 1.0f;
    shadowAttPRD.seed = prd.seed;
    uint32_t u0, u1;
    packPointer( &shadowAttPRD, u0, u1 );  
  
    // trace shadow ray
    int squaredShadowRays = 1;
    float shadowTotal = 0.0f;
    for (int i = 0; i < squaredShadowRays; ++i) {
        for (int j = 0; j < squaredShadowRays; ++j) {

            //uint32_t seed = tea<4>(  , i * squaredShadowRays + j );

            //const float2 subpixel_jitter = make_float2( i * delta.x + delta.x *  rnd( seed ), j * delta.y + delta.y * rnd( seed ) );
            //const float2 subpixel_jitter = make_float2( rnd( seed )-0.5f, rnd( seed )-0.5f );
            lPos.x = -0.2 + i * 1.0/squaredShadowRays * 0.4f + rnd(prd.seed) * 1.0/squaredShadowRays * 0.4;
            lPos.z = -0.2 + j * 1.0/squaredShadowRays * 0.4f + rnd(prd.seed) * 1.0/squaredShadowRays * 0.4;
            lDir = normalize(lPos - pos);
            optixTrace(optixLaunchParams.traversable,
                pos,
                lDir,
                0.00001f,           // tmin
                10,                 // tmax
                0.0f,               // rayTime
                OptixVisibilityMask( 255 ),
                OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
                SHADOW,             // SBT offset
                RAY_TYPE_COUNT,     // SBT stride
                SHADOW,             // missSBTIndex 
                u0, u1 );

                shadowTotal += shadowAttPRD.shadowAtt;
        }
    }
    shadowTotal /= (squaredShadowRays * squaredShadowRays);

    if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {  
        // get barycentric coordinates
        // compute pixel texture coordinate
        const float4 tc
          = (1.f-u-v) * sbtData.vertexD.texCoord0[index.x]
          +         u * sbtData.vertexD.texCoord0[index.y]
          +         v * sbtData.vertexD.texCoord0[index.z];
        // fetch texture value
        
        float4 fromTexture = tex2D<float4>(sbtData.texture,tc.x,tc.y);
        prd.color = make_float3(fromTexture) * min(intensity * shadowTotal + 0.0, 1.0);
    }
    else
        prd.color = sbtData.color * min(intensity * shadowTotal + 0.0, 1.0);
}


// any hit to ignore intersections with back facing geometry
extern "C" __global__ void __anyhit__radiance() {

}


// miss sets the background color
extern "C" __global__ void __miss__radiance() {

    colorPRD &prd = *(colorPRD*)getPRD<colorPRD>();
    // set blue as background color
    prd.color = make_float3(0.0f, 0.0f, 1.0f);
}


// -----------------------------------------------
// Shadow rays

extern "C" __global__ void __closesthit__shadow() {

    shadowPRD &prd = *(shadowPRD*)getPRD<shadowPRD>();
    prd.shadowAtt = 0.0f;
}


// any hit for shadows
extern "C" __global__ void __anyhit__shadow() {

}


// miss for shadows
extern "C" __global__ void __miss__shadow() {

    shadowPRD &prd = *(shadowPRD*)getPRD<shadowPRD>();
    prd.shadowAtt = 1.0f;
}




// -----------------------------------------------
// Primary Rays

extern "C" __global__ void __raygen__renderFrame() {

    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const auto &camera = optixLaunchParams.camera;  
    
	if (optixLaunchParams.frame.frame == 0 && ix == 0 && iy == 0) {
		// print info to console
		printf("===========================================\n");
        printf("Nau Ray-Tracing Debug\n");
        const float4 &ld = optixLaunchParams.global->lightPos;
        printf("LightPos: %f, %f %f %f\n", ld.x,ld.y,ld.z,ld.w);
        printf("Launch dim: %u %u\n", optixGetLaunchDimensions().x, optixGetLaunchDimensions().y);
        printf("Rays per pixel squared: %d \n", optixLaunchParams.frame.raysPerPixel);
		printf("===========================================\n");
	}

    float lensDistance  = optixLaunchParams.global->lensDistance;
    float focalDistance = optixLaunchParams.global->focalDistance;
    float aperture = optixLaunchParams.global->aperture;
    float3 frente = normalize(cross(camera.vertical,camera.horizontal));
    float3 lensCentre = camera.position + frente*lensDistance;

    // ray payload
    colorPRD pixelColorPRD;
    pixelColorPRD.color = make_float3(1.f);

    float raysPerPixel = float(optixLaunchParams.frame.raysPerPixel);
    // half pixel
    float2 delta = make_float2(1.0f/raysPerPixel, 1.0f/raysPerPixel);

    // compute ray direction
    // normalized screen plane position, in [-1, 1]^2
  
    float red = 0.0f, blue = 0.0f, green = 0.0f;
    for (int i = 0; i < raysPerPixel; ++i) {
        for (int j = 0; j < raysPerPixel; ++j) {

            uint32_t seed = tea<4>( ix * optixGetLaunchDimensions().x + iy, i*raysPerPixel + j );

            pixelColorPRD.seed = seed;
            uint32_t u0, u1;
            packPointer( &pixelColorPRD, u0, u1 );  
            //const float2 subpixel_jitter = make_float2( i * delta.x + delta.x *  rnd( seed ), j * delta.y + delta.y * rnd( seed ) );
            //const float2 subpixel_jitter = make_float2( rnd( seed )-0.5f, rnd( seed )-0.5f );
            const float2 subpixel_jitter = make_float2(i * delta.x, j * delta.y);
            const float2 screen(make_float2(ix + subpixel_jitter.x, iy + subpixel_jitter.y)
                            / make_float2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y) * 2.0 - 1.0);
        
        // note: nau already takes into account the field of view and ratio when computing 
        // camera horizontal and vertival

            float3 cPos = camera.position+(-screen.x)*camera.horizontal + (-screen.y ) * camera.vertical;

            float3 rayDir = normalize(lensCentre - cPos);

            //float3 proj_frente_rayDir = (dot(rayDir,frente)/(length(frente)*length(frente)))*frente; //frente ta normalizado POSSO SIMPLIFICAR
            float3 proj_frente_rayDir = dot(rayDir,frente)*frente;


            // Vetor que vai do centro da lente para o ponto de foco no plano de foco
            float3 ray = rayDir * focalDistance / length(proj_frente_rayDir);

            float3 pFocal = lensCentre + ray;


            float randR = aperture * sqrt(rnd(seed));

            float randA = rnd(seed) * 2 * M_PIf;

            float x = randR * cos(randA);
            float y = randR * sin(randA);

            float3 randAperture = lensCentre + camera.horizontal * x + camera.vertical * y;//make_float3(((lensCentre.x + camera.horizontal * x),(lensCentre.y + camera.vertical * y),lensCentre.z));

            float3 rayDirection = pFocal - randAperture;
            
            // trace primary ray
            optixTrace(optixLaunchParams.traversable,
                    randAperture,
                    rayDirection,
                    0.f,    // tmin
                    1e20f,  // tmax
                    0.0f,   // rayTime
                    OptixVisibilityMask( 255 ),
                    OPTIX_RAY_FLAG_NONE,//,OPTIX_RAY_FLAG_DISABLE_ANYHIT
                    PHONG,             // SBT offset
                    RAY_TYPE_COUNT,               // SBT stride
                    PHONG,             // missSBTIndex 
                    u0, u1 );

            red += pixelColorPRD.color.x / (raysPerPixel*raysPerPixel);
            green += pixelColorPRD.color.y / (raysPerPixel*raysPerPixel);
            blue += pixelColorPRD.color.z / (raysPerPixel*raysPerPixel);
        }
    }

    //convert float (0-1) to int (0-255)
    const int r = int(255.0f*red);
    const int g = int(255.0f*green);
    const int b = int(255.0f*blue);
    // convert to 32-bit rgba value 
    const uint32_t rgba = 0xff000000
      | (r<<0) | (g<<8) | (b<<16);
    // compute index
    const uint32_t fbIndex = ix + iy*optixGetLaunchDimensions().x;
    // write to output buffer
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}
